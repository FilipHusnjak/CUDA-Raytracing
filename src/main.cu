#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <fstream>
#include <chrono>

#define MAX_F 3.4e+38f

#define BLOCK_SIZE 16

#define BYTES_PER_PIXEL 3

#define WIDTH 4096
#define HEIGHT 4096

#define NUM_SPHERES 20

#define MAX_DEPTH 3

using byte = unsigned char;

struct vec3 {
    float x;
    float y;
    float z;

    [[nodiscard]] __device__ __host__ float norm() const {
        return std::sqrt(x * x + y * y + z * z);
    }

    __device__ __host__ void normalize() {
        float n = norm();
        x /= n;
        y /= n;
        z /= n;
    }

    [[nodiscard]] __device__ __host__ float dot(const vec3 &other) const {
        return x * other.x + y * other.y + z * other.z;
    }

    __device__ __host__ vec3 operator+(const vec3 &other) const {
        return {x + other.x, y + other.y, z + other.z};
    }

    __device__ __host__ vec3 operator-(const vec3 &other) const {
        return {x - other.x, y - other.y, z - other.z};
    }

    __device__ __host__ vec3 operator*(float v) const {
        return {x * v, y * v, z * v};
    }
};

struct ray {
    vec3 origin;
    vec3 dir;
};

struct sphere {
    vec3 color;
    vec3 center;
    float r;
};

__device__ __host__ float gpu_min(float a, float b) {
    return a < b ? a : b;
}

__device__ __host__ float gpu_max(float a, float b) {
    return a > b ? a : b;
}

void write_bmp_image(int width, int height, const byte *img, const std::string &filename) {
    std::ofstream ofs;
    ofs.open(filename, std::ios::binary | std::ios::out);
    if (!ofs) throw std::runtime_error("Unable to open file!");
    byte bmp_file_header[14] = {'B', 'M', 0, 0, 0, 0, 0, 0, 0, 0, 54, 0, 0, 0};
    byte bmp_info_header[40] = {40, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, BYTES_PER_PIXEL * 8, 0};
    int file_size = 54 + BYTES_PER_PIXEL * width * height;
    bmp_file_header[2] = (byte) (file_size);
    bmp_file_header[3] = (byte) (file_size >> 8);
    bmp_file_header[4] = (byte) (file_size >> 16);
    bmp_file_header[5] = (byte) (file_size >> 24);
    bmp_info_header[4] = (byte) (width);
    bmp_info_header[5] = (byte) (width >> 8);
    bmp_info_header[6] = (byte) (width >> 16);
    bmp_info_header[7] = (byte) (width >> 24);
    bmp_info_header[8] = (byte) (height);
    bmp_info_header[9] = (byte) (height >> 8);
    bmp_info_header[10] = (byte) (height >> 16);
    bmp_info_header[11] = (byte) (height >> 24);
    ofs.write((char *) bmp_file_header, sizeof(bmp_file_header));
    ofs.write((char *) bmp_info_header, sizeof(bmp_info_header));
    ofs.write((char *) img, BYTES_PER_PIXEL * width * height);
    ofs.close();
}

__device__ __host__ bool intersect(const ray &r, const sphere &s, float &t) {
    float a = r.dir.dot(r.dir);
    vec3 l = r.origin - s.center;
    float b = 2 * r.dir.dot(l);
    float c = l.dot(l) - s.r * s.r;
    float d = b * b - 4 * a * c;
    if (d <= 0) return false;
    float d_sqrt = std::sqrt(d);
    float q = b < 0 ? (-b - d_sqrt) / 2.f : (-b + d_sqrt) / 2.f;
    float t0 = q / a;
    float t1 = c / q;
    float t_max = gpu_max(t0, t1);
    float t_min = gpu_min(t0, t1);
    if (t_max <= 0.f) return false;
    t = t_min >= 0 ? t_min : t_max;
    return true;
}

__device__ __host__ vec3 trace_ray(const ray &r, const sphere *spheres, int depth) {
    vec3 light_pos = {2.f, 2.f, -3.f};
    vec3 light_color = {1.f, 1.f, 1.f};
    float diff_power = 1.f;
    float spec_power = 1.f;
    float t_min = MAX_F;
    sphere s{};
    for (int i = 0; i < NUM_SPHERES; i++) {
        float t;
        if (!intersect(r, spheres[i], t)) continue;
        if (t < t_min) {
            t_min = t;
            s = spheres[i];
        }
    }
    if (t_min == MAX_F) return {0.f, 0.f, 0.f};
    vec3 ambient = s.color * 0.2f;
    vec3 point = r.dir * t_min + r.origin;
    vec3 n = point - s.center;
    n.normalize();
    vec3 light_dir = light_pos - point;
    light_dir.normalize();
    float n_dot_l = n.dot(light_dir);
    float intensity = gpu_max(n_dot_l, 0.f);
    vec3 diffuse = s.color * intensity * diff_power;
    vec3 h = light_dir - r.dir;
    h.normalize();
    float n_dot_h = n.dot(h);
    intensity = std::pow(gpu_max(n_dot_h, 0.f), 50.f);
    vec3 specular = light_color * intensity * spec_power;
    vec3 color = ambient + diffuse + specular;
    if (depth >= MAX_DEPTH) return color;
    vec3 refl_dir = r.dir - n * 2 * r.dir.dot(n);
    refl_dir.normalize();
    ray refl_r = {point + refl_dir * 0.01f, refl_dir};
    vec3 refl_color = trace_ray(refl_r, spheres, depth + 1);
    // Fresnel-Schlick approximation
    float f0 = 0.5f;
    float f = f0 + (1 - f0) * std::pow(1 + h.dot(r.dir), 5.f);
    return refl_color * f + color * (1.f - f);
}

__device__ __host__ vec3 trace(int i, int j, const sphere *spheres) {
    auto dist = (float) WIDTH;
    vec3 dir = {(float) WIDTH / 2.f - (float) j, (float) i - (float) HEIGHT / 2.f, dist};
    dir.normalize();
    ray r = {{0.f, 0.f, -1.f}, dir};
    return trace_ray(r, spheres, 0);
}

__device__ __host__ int clamp(float value) {
    return value < 0.f ? 0 : (value > 255.f ? 255 : (int) value);
}

__global__ void gpu_trace(byte *img, const sphere *spheres) {
    const unsigned int j = threadIdx.x + BLOCK_SIZE * blockIdx.x;
    const unsigned int i = threadIdx.y + BLOCK_SIZE * blockIdx.y;
    vec3 color = trace((int) i, (int) j, spheres);
    img[(i * WIDTH + j) * BYTES_PER_PIXEL] = (byte) clamp(color.z * 255.f);
    img[(i * WIDTH + j) * BYTES_PER_PIXEL + 1] = (byte) clamp(color.y * 255.f);
    img[(i * WIDTH + j) * BYTES_PER_PIXEL + 2] = (byte) clamp(color.x * 255.f);
}

float random() {
    return (float) std::rand() / RAND_MAX;
}

int main() {
    srand(time(nullptr));
    sphere spheres[NUM_SPHERES];
    for (auto &sphere: spheres) {
        sphere = {{random(), random(), random()}, {random() * 6.f - 3.f, random() * 6.f - 3.f, random() * 4.f + 6.f},
                  random() + 0.4f};
    }
    auto t1 = std::chrono::system_clock::now();
    byte *img = new byte[WIDTH * HEIGHT * BYTES_PER_PIXEL];
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            vec3 color = trace(i, j, spheres);
            img[(i * WIDTH + j) * BYTES_PER_PIXEL] = (byte) clamp(color.z * 255.f);
            img[(i * WIDTH + j) * BYTES_PER_PIXEL + 1] = (byte) clamp(color.y * 255.f);
            img[(i * WIDTH + j) * BYTES_PER_PIXEL + 2] = (byte) clamp(color.x * 255.f);
        }
    }
    auto t2 = std::chrono::system_clock::now();
    printf("CPU time: %f ms\n", std::chrono::duration<float, std::chrono::milliseconds::period>(t2 - t1).count());
    write_bmp_image(WIDTH, HEIGHT, img, "result.bmp");

    byte *d_img;
    t1 = std::chrono::system_clock::now();
    hipMalloc(&d_img, WIDTH * HEIGHT * BYTES_PER_PIXEL);
    sphere * d_spheres;
    hipMalloc(&d_spheres, NUM_SPHERES * sizeof(sphere));
    hipMemcpy(d_spheres, spheres, NUM_SPHERES * sizeof(sphere), hipMemcpyHostToDevice);
    dim3 grid = dim3((WIDTH + (BLOCK_SIZE - 1)) / BLOCK_SIZE, (HEIGHT + (BLOCK_SIZE - 1)) / BLOCK_SIZE);
    dim3 block = dim3(BLOCK_SIZE, BLOCK_SIZE);
    gpu_trace<<<grid, block>>>(d_img, d_spheres);
    byte *img_gpu = new byte[WIDTH * HEIGHT * BYTES_PER_PIXEL];
    hipMemcpy(img_gpu, d_img, WIDTH * HEIGHT * BYTES_PER_PIXEL, hipMemcpyDeviceToHost);
    t2 = std::chrono::system_clock::now();
    printf("GPU time: %f ms\n", std::chrono::duration<float, std::chrono::milliseconds::period>(t2 - t1).count());
    write_bmp_image(WIDTH, HEIGHT, img_gpu, "result_gpu.bmp");

    delete[] img;
    delete[] img_gpu;
    hipFree(d_spheres);
    hipFree(d_img);
}
